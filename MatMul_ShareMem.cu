#include "hip/hip_runtime.h"
typedef struct {
    int width;
    int height;
    int stride;
    float* element;
} Matrix;

__device__ float GetElement(const Matrix A, int row, int col)
{
    float res = A.element[row * A.width + col];
    return res;
}

__device__ void SetElement(Matrix A, int row, int col, float value)
{
    A.element[row * A.width + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix sub;
    sub.height = sub.width = BLOCK_SIZE;
    sub.stride = A.stride;
    sub.element = &A.element[A.width * BLOCK_SIZE * row
                                     + BLOCK_SIZE * col];
}

#define BLOCK_SIZE 16

__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    d_A.stride = A.width;
    size_t size = d_A.width * d_A.height * sizeof(float);
    hipMalloc(&d_A.element, size);
    hipMemcpy(d_A.element, A.element, size, cudaMemcpyHostToDevic);

    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    d_B.stride = B.width;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.element, size);
    hipMemcpy(d_B.element, B.element, size, hipMemcpyHostToDevice);

    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    d_C.stride = C.stride;
    size = d_C.width * d_C.height * sizeof(float);
    hipMalloc(&d_C.element, size);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width/dimBlock.x, A.height/dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C.element, d_C.element, size, hipMemcpyDeviceToHost);

    hipFree(d_A.element);
    hipFree(d_B.element);
    hipFree(d_C.element);


}

__global__ MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    int subRow = threadIdx.y;
    int subCol = threadIdx.x;

    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    float Cval = 0;

    for(int i = 0; i < A.width/BLOCK_SIZE; i++)
    {
        Matrix Asub = GetSubMatrix(A, blockRow, i);
        Matrix Bsub = GetSubMatrix(B, i, blockCol);

        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[subRow][subCol] = GetElement(Asub, subRow, subCol);
        Bs[subRow][subCol] = GetElement(Bsub, subRow, subCol);

        __syncthreads();
        for(int j = 0; j < BLOCK_SIZE; j++)
        {
            Cval += As[subRow][j] * Bs[j][subCol];
        }
        __syncthreads();
    }
    SetElement(Csub, subRow, subCol, Cval);
}